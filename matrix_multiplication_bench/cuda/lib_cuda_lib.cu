#include <hipblas.h>
#include "../benchmark_library.h"


void init(GraficObject *device_object, char* device_name){
    init(device_object, 0,0, device_name);
}

void init(GraficObject *device_object, int platform ,int device, char* device_name){
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    //printf("Using device: %s\n", prop.name);
    strcpy(device_name,prop.name);
    //event create 
    device_object->start = new hipEvent_t;
    device_object->stop = new hipEvent_t;
    device_object->start_memory_copy_device = new hipEvent_t;
    device_object->stop_memory_copy_device = new hipEvent_t;
    device_object->start_memory_copy_host = new hipEvent_t;
    device_object->stop_memory_copy_host= new hipEvent_t;
    
    hipEventCreate(device_object->start);
    hipEventCreate(device_object->stop);
    hipEventCreate(device_object->start_memory_copy_device);
    hipEventCreate(device_object->stop_memory_copy_device);
    hipEventCreate(device_object->start_memory_copy_host);
    hipEventCreate(device_object->stop_memory_copy_host);
}


bool device_memory_init(GraficObject *device_object, unsigned int size_a_matrix, unsigned int size_b_matrix, unsigned int size_c_matrix){

   // Allocate the device input vector A
	hipError_t err = hipSuccess;
    err = hipMalloc((void **)&device_object->d_A, size_a_matrix * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }

    // Allocate the device input vector B
    err = hipMalloc((void **)&device_object->d_B, size_b_matrix * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }

    // Allocate the device output vector C
    err = hipMalloc((void **)&device_object->d_C, size_c_matrix * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    return true;
}

void copy_memory_to_device(GraficObject *device_object, bench_t* h_A, bench_t* h_B, unsigned int size_a, unsigned int size_b){
    hipEventRecord(*device_object->start_memory_copy_device);
	hipError_t err = hipMemcpy(device_object->d_A, h_A, sizeof(bench_t) * size_a, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(device_object->d_B, h_B, sizeof(bench_t) * size_b, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    hipEventRecord(*device_object->stop_memory_copy_device);   
}
void execute_kernel(GraficObject *device_object, unsigned int n, unsigned int m,unsigned int w){
    // cublas settings
    int lda=m,ldb=m,ldc=m;
    const bench_t alf = 1;
    const bench_t bet = 0;
    const bench_t *alpha = &alf;
    const bench_t *beta = &bet;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEventRecord(*device_object->start);
    //hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    #ifdef INT
    printf("CUBLAS NOT SUPPORT INT OPERATIOS\n");
    #elif FLOAT
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, w, alpha, device_object->d_B, lda, device_object->d_A, ldb, beta, device_object->d_C, ldc);
    #else 
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, w, alpha, device_object->d_B, lda, device_object->d_A, ldb, beta, device_object->d_C, ldc);
    #endif
    
    hipEventRecord(*device_object->stop);
    // destroy cublas
    hipblasDestroy(handle);
}

void copy_memory_to_host(GraficObject *device_object, bench_t* h_C, int size){
    hipEventRecord(*device_object->start_memory_copy_host);
    hipMemcpy(h_C, device_object->d_C, size * sizeof(bench_t), hipMemcpyDeviceToHost);
    hipEventRecord(*device_object->stop_memory_copy_host);
    }

float get_elapsed_time(GraficObject *device_object, bool csv_format){
    hipEventSynchronize(*device_object->stop_memory_copy_host);
    float milliseconds_h_d = 0, milliseconds = 0, milliseconds_d_h = 0;
    // memory transfer time host-device
    hipEventElapsedTime(&milliseconds_h_d, *device_object->start_memory_copy_device, *device_object->stop_memory_copy_device);
    // kernel time
    hipEventElapsedTime(&milliseconds, *device_object->start, *device_object->stop);
    //  memory transfer time device-host
    hipEventElapsedTime(&milliseconds_d_h, *device_object->start_memory_copy_host, *device_object->stop_memory_copy_host);
    
    if (csv_format){
         printf("%.10f;%.10f;%.10f;\n", milliseconds_h_d,milliseconds,milliseconds_d_h);
    }else{
         printf("Elapsed time Host->Device: %.10f miliseconds\n", milliseconds_h_d);
         printf("Elapsed time kernel: %.10f miliseconds\n", milliseconds);
         printf("Elapsed time Device->Host: %.10f miliseconds\n", milliseconds_d_h);
    }
    return milliseconds;
}

void clean(GraficObject *device_object){
	hipError_t err = hipSuccess;
	err = hipFree(device_object->d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return;
    }

    err = hipFree(device_object->d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return;
    }


    // delete events
    delete device_object->start;
    delete device_object->stop;
    delete device_object->start_memory_copy_device;
    delete device_object->stop_memory_copy_device;
    delete device_object->start_memory_copy_host;
    delete device_object->stop_memory_copy_host;
}
