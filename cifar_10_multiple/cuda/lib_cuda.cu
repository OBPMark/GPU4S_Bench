#include "hip/hip_runtime.h"
#include "../benchmark_library.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
//#define BLOCK_SIZE 32
__global__ void
covolution_kernel(const bench_t *A, bench_t *B, const bench_t *kernel,const int n, const int m, const int w, const int kernel_size)
{
    unsigned int size = n;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int kernel_rad = kernel_size / 2;

    bench_t sum = 0;

    if (x < size && y < size)
    {
        for(int i = -kernel_rad; i <= kernel_rad; ++i) // loop over kernel_rad  -1 to 1 in kernel_size 3 
            {
                for(int j = -kernel_rad; j <= kernel_rad; ++j){
                    // get value
                    bench_t value = 0;
                    
                    if (i + x < 0 || j + y < 0)
                    {
                        value = 0;
                        //printf("ENTRO %d %d\n", i + x , j + y);
                    }
                    else if ( i + x > size - 1 || j + y > size -1 )
                    {
                        value = 0;
                        //printf("ENTRO UPPER%d %d\n", i + x , j + y);
                    }
                    else
                    {
                        value = A[(x + i)*size+(y + j)];
                    }
                    
                   
                    sum += value * kernel[(i+kernel_rad)* kernel_size + (j+kernel_rad)];
                }
            }
    
    B[x*size+y ] = sum;
    }
    
}
__global__ void
relu_kernel(const bench_t *A, bench_t *B, const int size)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    bench_t threshold = 0;
    if (i < size && j < size){
        #ifdef INT
        B[i*size+j] = max(threshold, A[i*size+j]);
        #elif FLOAT
        B[i*size+j] = max(threshold, A[i*size+j]);
        #else
        B[i*size+j] = fmax(threshold, A[i*size+j]);
        #endif
    }
}

__global__ void
relu_linear_kernel(const bench_t *A, bench_t *B, const int size)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    bench_t threshold = 0;
    if (i  < size){
        
        #ifdef INT
        B[i] = max(threshold, A[i]);
        #elif FLOAT
        B[i] = max(threshold, A[i]);
        #else
        B[i] = fmax(threshold, A[i]);
        
        #endif
    }
}

__global__ void
max_pooling_kernel(const bench_t *A, bench_t *B, const int size, const unsigned int stride,  const unsigned int lateral_stride)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
   
    if (i < size && j < size){
        bench_t max_value = A[((i * stride)) * size + ((j*stride))];
        for(unsigned int x = 0; x < stride; ++x)
        {
            for(unsigned int y = 0; y < stride; ++y)
            {
                //printf("max %f, value %f, pos x %d, pos y %d \n", max_value, A[(i + x) * size + (j +y)],i + x , j +y);
                max_value = max(max_value, A[((i * stride) + x) * size + ((j*stride) +y)]);
                
            }
        }
        //printf("value %f, position %d, lateral_stride %d\n", max_value,i * lateral_stride + j, lateral_stride );
        B[i * lateral_stride + j ] = max_value;
    }
}
__global__ void
lrn_kernel(const bench_t *A, bench_t *B, const int size)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size && j < size){
        #ifdef INT
        B[i*size+j] = A[i*size+j]/powf((K+ALPHA*powf(A[i*size+j],2)),BETA);
        #elif FLOAT
        B[i*size+j] = A[i*size+j]/powf((K+ALPHA*powf(A[i*size+j],2)),BETA);
        #else
        B[i*size+j] = A[i*size+j]/powf((K+ALPHA*powf(A[i*size+j],2)),BETA);
        #endif
    }
}

__global__ void
matrix_multiplication_kernel(const bench_t *A,const bench_t *B,  bench_t *C, const int n, const int m, const int w)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < m){
        bench_t acumulated = 0;
        for (unsigned int k_d = 0; k_d < w; ++k_d )
        {   
            //printf("position %d valor %f, k_d %d , i %d, j %d, n %d\n ", i*n+k_d, acumulated,k_d,w,i,n);
            acumulated += A[i*w+k_d] * B[k_d*m +j];
        }
       
        //printf("value %f position %d \n", acumulated,i *m + j);
        C[i*m+j] =  acumulated;
    }
}

__global__ void
softmax_kernel(const bench_t *A, bench_t *B, bench_t *sum_d_B,const int size)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size && j < size){
        #ifdef INT
        B[i*size+j] = exp(A[i*size+j]);
        #elif FLOAT
        B[i*size+j] = expf(A[i*size+j]);
        #else
        B[i*size+j] = exp(A[i*size+j]);
        #endif
        atomicAdd(sum_d_B, B[i*size+j]);
    }
}
__global__ void
softmax_finish_kernel(bench_t *B, bench_t *sum_d_B,const int size)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size && j < size){
        B[i*size+j] = (B[i*size+j]/(*sum_d_B));
    }
}

//////////////////////////////////////////////////////////////////////////////////////
// End CUDA part
//////////////////////////////////////////////////////////////////////////////////////


void init(GraficObject *device_object, char* device_name){
	init(device_object, 0,0, device_name);
}

void init(GraficObject *device_object, int platform ,int device, char* device_name){
	hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	//printf("Using device: %s\n", prop.name);
    strcpy(device_name,prop.name);
    //event create 
    device_object->start = new hipEvent_t;
    device_object->stop = new hipEvent_t;
    device_object->start_memory_copy_device = new hipEvent_t;
    device_object->stop_memory_copy_device = new hipEvent_t;
    device_object->start_memory_copy_host = new hipEvent_t;
    device_object->stop_memory_copy_host= new hipEvent_t;
    
    hipEventCreate(device_object->start);
    hipEventCreate(device_object->stop);
    hipEventCreate(device_object->start_memory_copy_device);
    hipEventCreate(device_object->stop_memory_copy_device);
    hipEventCreate(device_object->start_memory_copy_host);
    hipEventCreate(device_object->stop_memory_copy_host);
}


bool device_memory_init(GraficObject *device_object, unsigned int input_data, unsigned int output_data, unsigned int kernel_1, unsigned int kernel_2, unsigned int stride_1, unsigned int stride_2, unsigned int neurons_dense_1, unsigned int neurons_dense_2, unsigned int number_of_images){
   // Allocate input
	hipError_t err = hipSuccess;
    err = hipMalloc((void **)&(device_object->input_data), number_of_images * input_data * input_data * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // Allocate kernel
    err = hipMalloc((void **)&(device_object->kernel_1), kernel_1 * kernel_1 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // Allocate conv 1 output
    err = hipMalloc((void **)&(device_object->conv_1_output), input_data * input_data * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // Allocate pooling output
    unsigned int size_pooling_1 = input_data / stride_1;
    err = hipMalloc((void **)&(device_object->pooling_1_output), size_pooling_1 * size_pooling_1 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // Allocate kernel 2
    err = hipMalloc((void **)&(device_object->kernel_2), kernel_2 * kernel_2 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // Allocate conv 1 output
    err = hipMalloc((void **)&(device_object->conv_2_output), size_pooling_1 * size_pooling_1 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // Allocate pooling output
    unsigned int size_pooling_2 = size_pooling_1 / stride_2;
    err = hipMalloc((void **)&(device_object->pooling_2_output), size_pooling_2 * size_pooling_2 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    //dense layer 1 weights 
    unsigned int weights_layer_1 = size_pooling_2 * size_pooling_2 * neurons_dense_1;

    err = hipMalloc((void **)&(device_object->dense_layer_1_weights), weights_layer_1* sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // dense layer output 1
    err = hipMalloc((void **)&(device_object->dense_layer_1_output), neurons_dense_1 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    //dense layer 2 weights 
    unsigned int weights_layer_2 = neurons_dense_1 * neurons_dense_2;
    err = hipMalloc((void **)&(device_object->dense_layer_2_weights), weights_layer_2  * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // dense layer output 2
    err = hipMalloc((void **)&(device_object->dense_layer_2_output), neurons_dense_2 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
     // sum data
    err = hipMalloc((void **)&(device_object->sum_ouput), sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    // output data
    err = hipMalloc((void **)&(device_object->output_data), number_of_images * neurons_dense_2 * sizeof(bench_t));

    if (err != hipSuccess)
    {
        return false;
    }
    return true;
 }

void copy_memory_to_device(GraficObject *device_object, bench_t* input_data, bench_t* kernel_1_data, bench_t* kernel_2_data, bench_t* weights_1 ,bench_t* weights_2,unsigned int input , unsigned int kernel_size_1, unsigned int kernel_size_2, unsigned int weights_1_size, unsigned int weights_2_size, unsigned int number_of_images){
    hipEventRecord(*device_object->start_memory_copy_device);
	hipError_t err = hipMemcpy(device_object->input_data, input_data, sizeof(bench_t) * input * input * number_of_images, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector input from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(device_object->kernel_1, kernel_1_data, sizeof(bench_t) * kernel_size_1 * kernel_size_1, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector kernel_1 from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(device_object->kernel_2, kernel_2_data, sizeof(bench_t) * kernel_size_2 * kernel_size_2, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector kernel_2 from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(device_object->dense_layer_1_weights, weights_1, sizeof(bench_t) * weights_1_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector weights_layer_1 from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(device_object->dense_layer_2_weights, weights_2, sizeof(bench_t) * weights_2_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector weights_layer_2 from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    hipMemset(device_object->sum_ouput, 0,  sizeof(bench_t));
    hipEventRecord(*device_object->stop_memory_copy_device);
    
}
void execute_kernel(GraficObject *device_object, unsigned int input_data, unsigned int output_data, unsigned int kernel_1, unsigned int kernel_2, unsigned int stride_1, unsigned int stride_2, unsigned int neurons_dense_1, unsigned int neurons_dense_2, unsigned int number_of_images){
    // execute net 
    
    hipEventRecord(*device_object->start);
    bench_t* aux_output_data = device_object->output_data;
    bench_t* aux_input_data = device_object->input_data;
    
    for(unsigned int position = 0; position < number_of_images; ++position)
    {
        aux_input_data = device_object->input_data + position * input_data * input_data;
        aux_output_data = device_object->output_data + position * output_data;
        // 1-1 step convolution
        dim3 dimBlock, dimGrid;
        dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
        dimGrid = dim3(ceil(float(input_data)/dimBlock.x), ceil(float(input_data)/dimBlock.y));
        covolution_kernel<<<dimGrid, dimBlock>>>(aux_input_data, device_object->conv_1_output, device_object->kernel_1, input_data, input_data, input_data, kernel_1);

        // 1-2 step activation
        relu_kernel<<<dimGrid, dimBlock>>>(device_object->conv_1_output, device_object->conv_1_output, input_data);
        // 1-3 step pooling
        unsigned int size_lateral_1 = input_data / stride_1;
        if(size_lateral_1 < BLOCK_SIZE)
        {
            dimBlock = dim3(size_lateral_1, size_lateral_1);
            dimGrid = dim3(1, 1);
        }
        else
        {
            dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
            dimGrid = dim3(ceil(((float(size_lateral_1) / stride_1 ))/dimBlock.x), ceil(((float(size_lateral_1) / stride_1 ))/dimBlock.y));
        }
        max_pooling_kernel<<<dimGrid, dimBlock>>>(device_object->conv_1_output, device_object->pooling_1_output, input_data, stride_1, size_lateral_1);

        // 1-4 normalization
        lrn_kernel<<<dimGrid, dimBlock>>>(device_object->pooling_1_output, device_object->pooling_1_output, size_lateral_1);

        // 2-1 step convolution
        covolution_kernel<<<dimGrid, dimBlock>>>(device_object->pooling_1_output, device_object->conv_2_output, device_object->kernel_2, size_lateral_1, size_lateral_1, size_lateral_1, kernel_2);

        // 2-2 step activation
        relu_kernel<<<dimGrid, dimBlock>>>(device_object->conv_2_output, device_object->conv_2_output, size_lateral_1);
        // 2-3 normalization
        lrn_kernel<<<dimGrid, dimBlock>>>(device_object->conv_2_output, device_object->conv_2_output, size_lateral_1);
        // 2-4 step pooling

        unsigned int size_lateral_2 = size_lateral_1 / stride_2;
        if(size_lateral_2 < BLOCK_SIZE)
        {
            dimBlock = dim3(size_lateral_2, size_lateral_2);
            dimGrid = dim3(1, 1);
        }
        else
        {
            dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
            dimGrid = dim3(ceil(((float(size_lateral_2) / stride_2 ))/dimBlock.x), ceil(((float(size_lateral_2) / stride_2 ))/dimBlock.y));
        }
        max_pooling_kernel<<<dimGrid, dimBlock>>>(device_object->conv_2_output, device_object->pooling_2_output, size_lateral_1, stride_2, size_lateral_2);
        // dense layer 1
        dimBlock = dim3(BLOCK_SIZE, 1);
        dimGrid = dim3(ceil(float(neurons_dense_1)/dimBlock.x), 1);
        matrix_multiplication_kernel<<<dimGrid, dimBlock>>>(device_object->dense_layer_1_weights, device_object->pooling_2_output,device_object->dense_layer_1_output,neurons_dense_1, 1, size_lateral_2*size_lateral_2);
        //activation layer dense 1
        dimBlock = dim3(BLOCK_SIZE);
        dimGrid = dim3(ceil(float(neurons_dense_1)/dimBlock.x));
        relu_linear_kernel<<<dimGrid, dimBlock>>>(device_object->dense_layer_1_output, device_object->dense_layer_1_output, neurons_dense_1);
        // dense layer 2
        dimBlock = dim3(BLOCK_SIZE, 1);
        dimGrid = dim3(ceil(float(neurons_dense_2)/dimBlock.x), 1);

        matrix_multiplication_kernel<<<dimGrid, dimBlock>>>(device_object->dense_layer_2_weights, device_object->dense_layer_1_output, device_object->dense_layer_2_output, neurons_dense_2, 1, neurons_dense_1);
        // activation layer dense 2
        dimBlock = dim3(BLOCK_SIZE);
        dimGrid = dim3(ceil(float(neurons_dense_2)/dimBlock.x));
        relu_linear_kernel<<<dimGrid, dimBlock>>>(device_object->dense_layer_2_output, device_object->dense_layer_2_output, neurons_dense_2);

        // softmax 
        dimBlock = dim3(1, BLOCK_SIZE);
        dimGrid = dim3(1, ceil(float(neurons_dense_2)/dimBlock.x));

        softmax_kernel<<<dimGrid, dimBlock>>>(device_object->dense_layer_2_output, aux_output_data, device_object->sum_ouput, neurons_dense_2);
        softmax_finish_kernel<<<dimGrid, dimBlock>>>(aux_output_data, device_object->sum_ouput, neurons_dense_2);
        hipMemset(device_object->sum_ouput, 0, sizeof(bench_t));
    }
    hipEventRecord(*device_object->stop);
}

void copy_memory_to_host(GraficObject *device_object, bench_t* h_C, int size, unsigned int number_of_images){
    hipEventRecord(*device_object->start_memory_copy_host);
    hipMemcpy(h_C, device_object->output_data, number_of_images * size * sizeof(bench_t), hipMemcpyDeviceToHost);
    //hipMemcpy(h_C, device_object->dense_layer_2_output, 10 * sizeof(bench_t), hipMemcpyDeviceToHost);
    hipEventRecord(*device_object->stop_memory_copy_host);
}

float get_elapsed_time(GraficObject *device_object, bool csv_format){
    hipEventSynchronize(*device_object->stop_memory_copy_host);
    float milliseconds_h_d = 0, milliseconds = 0, milliseconds_d_h = 0;
    // memory transfer time host-device
    hipEventElapsedTime(&milliseconds_h_d, *device_object->start_memory_copy_device, *device_object->stop_memory_copy_device);
    // kernel time
    hipEventElapsedTime(&milliseconds, *device_object->start, *device_object->stop);
    //  memory transfer time device-host
    hipEventElapsedTime(&milliseconds_d_h, *device_object->start_memory_copy_host, *device_object->stop_memory_copy_host);
    
    if (csv_format){
         printf("%.10f;%.10f;%.10f;\n", milliseconds_h_d,milliseconds,milliseconds_d_h);
    }else{
         printf("Elapsed time Host->Device: %.10f miliseconds\n", milliseconds_h_d);
         printf("Elapsed time kernel: %.10f miliseconds\n", milliseconds);
         printf("Elapsed time Device->Host: %.10f miliseconds\n", milliseconds_d_h);
    }
    return milliseconds;
}

void clean(GraficObject *device_object){
    hipError_t err = hipSuccess;

    err = hipFree(device_object->input_data);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector input_data (error code %s)!\n", hipGetErrorString(err));
        return;
    }

    err = hipFree(device_object->kernel_1);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector kernel_1 (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->conv_1_output);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector conv_1_output (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->pooling_1_output);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector pooling_1_output (error code %s)!\n", hipGetErrorString(err));
        return;
    }

    err = hipFree(device_object->kernel_2);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector kernel_2 (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->conv_2_output);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector conv_2_output (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->pooling_2_output);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector pooling_2_output (error code %s)!\n", hipGetErrorString(err));
        return;
    }

    err = hipFree(device_object->dense_layer_1_weights);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector dense_layer_1_weights (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->dense_layer_2_weights);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector dense_layer_2_weights (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->dense_layer_1_output);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector dense_layer_1_output (error code %s)!\n", hipGetErrorString(err));
        return;
    }

    err = hipFree(device_object->dense_layer_2_output);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector dense_layer_2_output (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->output_data);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector output_data (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->sum_ouput);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector sum_ouput (error code %s)!\n", hipGetErrorString(err));
        return;
    }


    // delete events
    delete device_object->start;
    delete device_object->stop;
    delete device_object->start_memory_copy_device;
    delete device_object->stop_memory_copy_device;
    delete device_object->start_memory_copy_host;
    delete device_object->stop_memory_copy_host;
}
