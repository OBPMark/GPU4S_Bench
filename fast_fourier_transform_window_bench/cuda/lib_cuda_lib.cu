#include "hip/hip_runtime.h"
#include "../benchmark_library.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
//#define BLOCK_SIZE 1024

__global__ void
binary_reverse_kernel(const bench_t *A, bench_t *B, const int64_t size, const int group, const int position_off)
{
    
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int position = 0;
    if (id < size)
    {   
        position = (__brev(id) >> (32 - group)) * 2;
        B[(position) + (size * 2 * position_off)] = A[(id *2) + position_off];
        B[position + 1 +  (size * 2 * position_off)] = A[(id *2 + 1) + position_off];
    }
}

__global__ void
fft_kernel( bench_t *B, const int loop, const int inner_loop,const bench_t wr, const bench_t wi, const int64_t size, const int64_t position_off)
{   
    bench_t tempr, tempi;
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i;
    unsigned int j;
    // get I
    i = id *(loop * 2 * 2) + 1 + (inner_loop * 2); 
    j=i+(loop * 2 );

    tempr = wr*B[j-1 + (size * 2 * position_off)] - wi*B[j+ (size * 2 * position_off)];
    tempi = wr * B[j+ (size * 2 * position_off)] + wi*B[j-1+ (size * 2 * position_off)];
    
    B[j-1+ (size * 2 * position_off)] = B[i-1+ (size * 2 * position_off)] - tempr;
    B[j+ (size * 2 * position_off)] = B[i+ (size * 2 * position_off)] - tempi;
    B[i-1+ (size * 2 * position_off)] += tempr;
    B[i+ (size * 2 * position_off)] += tempi;
    
}

void init(GraficObject *device_object, char* device_name){
    init(device_object, 0,0, device_name);
}

void init(GraficObject *device_object, int platform ,int device, char* device_name){
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    //printf("Using device: %s\n", prop.name);
    strcpy(device_name,prop.name);
    //event create 
    device_object->start = new hipEvent_t;
    device_object->stop = new hipEvent_t;
    device_object->start_memory_copy_device = new hipEvent_t;
    device_object->stop_memory_copy_device = new hipEvent_t;
    device_object->start_memory_copy_host = new hipEvent_t;
    device_object->stop_memory_copy_host= new hipEvent_t;
    
    hipEventCreate(device_object->start);
    hipEventCreate(device_object->stop);
    hipEventCreate(device_object->start_memory_copy_device);
    hipEventCreate(device_object->stop_memory_copy_device);
    hipEventCreate(device_object->start_memory_copy_host);
    hipEventCreate(device_object->stop_memory_copy_host);
}


bool device_memory_init(GraficObject *device_object,  int64_t size_a_array, int64_t size_b_array){
    hipError_t err = hipSuccess;
    // Allocate the device input vector B
    err = hipMalloc((void **)&device_object->d_A, (size_a_array /2) * sizeof(bench_cuda_complex));

    if (err != hipSuccess)
    {
        return false;
    }
    err = hipMalloc((void **)&device_object->d_B, (size_b_array /2) * sizeof(bench_cuda_complex));

    if (err != hipSuccess)
    {
        return false;
    }
    return true;
}

void copy_memory_to_device(GraficObject *device_object, bench_t* h_A,int64_t size){
    hipError_t err = hipSuccess;
    bench_cuda_complex *h_signal = (bench_cuda_complex *)malloc(sizeof(bench_cuda_complex) * (size/2));
    for (unsigned int i = 0; i < (size/2); ++i){
        h_signal[i].x = h_A[i * 2];
        h_signal[i].y = h_A[i * 2 + 1];
    }

    hipEventRecord(*device_object->start_memory_copy_device);
    err = hipMemcpy(device_object->d_A, h_signal, sizeof(bench_cuda_complex) * (size / 2), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    hipEventRecord(*device_object->stop_memory_copy_device);

}

void aux_execute_kernel(GraficObject *device_object, int64_t size, bench_cuda_complex *d_A, bench_cuda_complex *d_B, hipfftHandle *plan){
    
    //bench_cuda_complex* d_B = device_object->d_B;
    
    #ifdef FLOAT
    hipfftPlan1d(plan, size/2, HIPFFT_C2C, 1);
    hipfftExecC2C(*plan, (bench_cuda_complex *)d_A, (bench_cuda_complex *)d_B, HIPFFT_FORWARD);
    #else 
    hipfftPlan1d(plan, size/2, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(*plan, (bench_cuda_complex *)d_A, (bench_cuda_complex *)d_B, HIPFFT_FORWARD);
    #endif
    
    
}

void execute_kernel(GraficObject *device_object, int64_t window, int64_t size){
    bench_cuda_complex* d_A = device_object->d_A;
    bench_cuda_complex* d_B = device_object->d_B;
    hipEventRecord(*device_object->start);
    hipfftHandle plan;
    for (unsigned int i = 0; i < (size * 2  - window + 1); i+=1){
        aux_execute_kernel(device_object, window, d_A, d_B, &plan);
        d_B += window;
        ++d_A;

    }
    hipfftDestroy(plan);
    hipEventRecord(*device_object->stop);
}

void copy_memory_to_host(GraficObject *device_object, bench_t* h_B, int64_t size){
    bench_cuda_complex *h_signal = (bench_cuda_complex *)malloc(sizeof(bench_cuda_complex) * (size/2));
    hipEventRecord(*device_object->start_memory_copy_host);
    hipMemcpy(h_signal, device_object->d_B, (size/2) * sizeof(bench_cuda_complex), hipMemcpyDeviceToHost);
    hipEventRecord(*device_object->stop_memory_copy_host);

    for (unsigned int i = 0; i < (size/2); ++i){
        h_B[i * 2] = h_signal[i].x;
        h_B[i * 2 + 1] = h_signal[i].y;
    }
}

float get_elapsed_time(GraficObject *device_object, bool csv_format){
    hipEventSynchronize(*device_object->stop_memory_copy_host);
    float milliseconds_h_d = 0, milliseconds = 0, milliseconds_d_h = 0;
    // memory transfer time host-device
    hipEventElapsedTime(&milliseconds_h_d, *device_object->start_memory_copy_device, *device_object->stop_memory_copy_device);
    // kernel time
    hipEventElapsedTime(&milliseconds, *device_object->start, *device_object->stop);
    //  memory transfer time device-host
    hipEventElapsedTime(&milliseconds_d_h, *device_object->start_memory_copy_host, *device_object->stop_memory_copy_host);
    
    if (csv_format){
         printf("%.10f;%.10f;%.10f;\n", milliseconds_h_d,milliseconds,milliseconds_d_h);
    }else{
         printf("Elapsed time Host->Device: %.10f miliseconds\n", milliseconds_h_d);
         printf("Elapsed time kernel: %.10f miliseconds\n", milliseconds);
         printf("Elapsed time Device->Host: %.10f miliseconds\n", milliseconds_d_h);
    }
    return milliseconds;
}

void clean(GraficObject *device_object){
    hipError_t err = hipSuccess;

     err = hipFree(device_object->d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(device_object->d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return;
    }


    // delete events
    delete device_object->start;
    delete device_object->stop;
    delete device_object->start_memory_copy_device;
    delete device_object->stop_memory_copy_device;
    delete device_object->start_memory_copy_host;
    delete device_object->stop_memory_copy_host;
}
