#include "../benchmark_library.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

void init(GraficObject *device_object, char* device_name){
    init(device_object, 0,0, device_name);
}

void init(GraficObject *device_object, int platform ,int device, char* device_name){
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    //printf("Using device: %s\n", prop.name);
    strcpy(device_name,prop.name);
    //event create 
    device_object->start = new hipEvent_t;
    device_object->stop = new hipEvent_t;
    device_object->start_memory_copy_device = new hipEvent_t;
    device_object->stop_memory_copy_device = new hipEvent_t;
    device_object->start_memory_copy_host = new hipEvent_t;
    device_object->stop_memory_copy_host= new hipEvent_t;
    
    hipEventCreate(device_object->start);
    hipEventCreate(device_object->stop);
    hipEventCreate(device_object->start_memory_copy_device);
    hipEventCreate(device_object->stop_memory_copy_device);
    hipEventCreate(device_object->start_memory_copy_host);
    hipEventCreate(device_object->stop_memory_copy_host);
}


bool device_memory_init(GraficObject *device_object, int64_t size_b_matrix){
    hipError_t err = hipSuccess;
    // Allocate the device input vector A
    err = hipMalloc((void **)&device_object->d_A, (size_b_matrix * size_b_matrix) * sizeof(bench_cuda_complex));

    if (err != hipSuccess)
    {
        return false;
    }
    err = hipMalloc((void **)&device_object->d_B, (size_b_matrix * size_b_matrix) * sizeof(bench_cuda_complex));

    if (err != hipSuccess)
    {
        return false;
    }
    return true;
}

void copy_memory_to_device(GraficObject *device_object, COMPLEX **h_B,int64_t size){
    hipError_t err = hipSuccess;
    bench_cuda_complex *h_signal = (bench_cuda_complex *)malloc(sizeof(bench_cuda_complex) * (size * size));
    for (unsigned int i = 0; i < (size); ++i){
         for (unsigned int j = 0; j < (size); ++j){
            h_signal[i * size + j].x = h_B[i][j].x;
            h_signal[i * size + j].y = h_B[i][j].y;
        }
    }

    hipEventRecord(*device_object->start_memory_copy_device);
    err = hipMemcpy(device_object->d_A, h_signal, sizeof(bench_cuda_complex) * (size * size), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    hipEventRecord(*device_object->stop_memory_copy_device);
    
}
void execute_kernel(GraficObject *device_object, int64_t size){

    hipfftHandle plan;

    hipEventRecord(*device_object->start);
    
    #ifdef FLOAT
    hipfftPlan2d(&plan, size, size, HIPFFT_C2C);
    hipfftExecC2C(plan, (hipfftComplex *)device_object->d_A, (hipfftComplex *)device_object->d_B, HIPFFT_FORWARD);
    #else 
    hipfftPlan2d(&plan, size, size, HIPFFT_Z2Z);
    hipfftExecZ2Z(plan, (hipfftDoubleComplex *)device_object->d_A, (hipfftDoubleComplex *)device_object->d_B, HIPFFT_FORWARD);
    #endif
    
    hipEventRecord(*device_object->stop);
    hipfftDestroy(plan);
    
}

void copy_memory_to_host(GraficObject *device_object, COMPLEX **h_B, int64_t size){
    bench_cuda_complex *h_signal = (bench_cuda_complex *)malloc(sizeof(bench_cuda_complex) * (size*size));
    hipEventRecord(*device_object->start_memory_copy_host);
    hipMemcpy(h_signal, device_object->d_B, (size*size) * sizeof(bench_cuda_complex), hipMemcpyDeviceToHost);
    hipEventRecord(*device_object->stop_memory_copy_host);
    for (unsigned int i = 0; i < (size); ++i){
         for (unsigned int j = 0; j < (size); ++j){
            h_B[i][j].x = h_signal[i * size + j].x;
            h_B[i][j].y = h_signal[i * size + j].y;
        }
    }
}

float get_elapsed_time(GraficObject *device_object, bool csv_format){
    hipEventSynchronize(*device_object->stop_memory_copy_host);
    float milliseconds_h_d = 0, milliseconds = 0, milliseconds_d_h = 0;
    // memory transfer time host-device
    hipEventElapsedTime(&milliseconds_h_d, *device_object->start_memory_copy_device, *device_object->stop_memory_copy_device);
    // kernel time
    hipEventElapsedTime(&milliseconds, *device_object->start, *device_object->stop);
    //  memory transfer time device-host
    hipEventElapsedTime(&milliseconds_d_h, *device_object->start_memory_copy_host, *device_object->stop_memory_copy_host);
    
    if (csv_format){
         printf("%.10f;%.10f;%.10f;\n", milliseconds_h_d,milliseconds,milliseconds_d_h);
    }else{
         printf("Elapsed time Host->Device: %.10f milliseconds\n", milliseconds_h_d);
         printf("Elapsed time kernel: %.10f milliseconds\n", milliseconds);
         printf("Elapsed time Device->Host: %.10f milliseconds\n", milliseconds_d_h);
    }
    return milliseconds;
}

void clean(GraficObject *device_object){
    hipError_t err = hipSuccess;

     err = hipFree(device_object->d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return;
    }

    err = hipFree(device_object->d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return;
    }


    // delete events
    delete device_object->start;
    delete device_object->stop;
    delete device_object->start_memory_copy_device;
    delete device_object->stop_memory_copy_device;
    delete device_object->start_memory_copy_host;
    delete device_object->stop_memory_copy_host;
}