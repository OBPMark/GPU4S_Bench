#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void filterData(const float *d_data,
                           const float *d_numerator, 
                           float *d_filteredData, 
                           const int numeratorLength,
                           const int filteredDataLength)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    float sum = 0.0f;

    if (i < filteredDataLength)
    {
        for (int j = 0; j < numeratorLength; j++)
        {
            // The first (numeratorLength-1) elements contain the filter state
            sum += d_numerator[j] * d_data[i + numeratorLength - j - 1];
        }
    }

    d_filteredData[i] = sum;
}

int main(void)
{
    // (Skipping error checks to make code more readable)

    int dataLength = 18042;
    int filteredDataLength = 16384;
    int numeratorLength= 1659;

    // Pointers to data, filtered data and filter coefficients
    // (Skipping how these are read into the arrays)
    float *h_data = new float[dataLength];
    float *h_filteredData = new float[filteredDataLength];
    float *h_filter = new float[numeratorLength];
    float *h_numerator = new float[numeratorLength];


    // Create device pointers
    float *d_data = nullptr;
    hipMalloc((void **)&d_data, dataLength * sizeof(float));

    float *d_numerator = nullptr;
    hipMalloc((void **)&d_numerator, numeratorLength * sizeof(float));

    float *d_filteredData = nullptr;
    hipMalloc((void **)&d_filteredData, filteredDataLength * sizeof(float));


    // Copy data to device
    hipMemcpy(d_data, h_data, dataLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_numerator, h_numerator, numeratorLength * sizeof(float), hipMemcpyHostToDevice);  

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (filteredDataLength + threadsPerBlock - 1) / threadsPerBlock;
    filterData<<<blocksPerGrid,threadsPerBlock>>>(d_data, d_numerator, d_filteredData, numeratorLength, filteredDataLength);

    // Copy results to host
    hipMemcpy(h_filteredData, d_filteredData, filteredDataLength * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_data);
    hipFree(d_numerator);
    hipFree(d_filteredData);

    // Do stuff with h_filteredData...
    for (int i= 0; i < filteredDataLength; ++i)
    {
        printf("%f ", h_filteredData[i]);
    }
    printf("\n");
    // Clean up some more
    delete [] h_data;
    delete [] h_filteredData;
    delete [] h_filter;
} 
