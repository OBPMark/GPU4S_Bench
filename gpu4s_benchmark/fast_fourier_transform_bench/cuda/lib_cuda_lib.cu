#include "../benchmark_library.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

void init(GraficObject *device_object, char* device_name){
    init(device_object, 0,0, device_name);
}

void init(GraficObject *device_object, int platform ,int device, char* device_name){
    hipSetDevice(device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    //printf("Using device: %s\n", prop.name);
    strcpy(device_name,prop.name);
    //event create 
    device_object->start = new hipEvent_t;
    device_object->stop = new hipEvent_t;
    device_object->start_memory_copy_device = new hipEvent_t;
    device_object->stop_memory_copy_device = new hipEvent_t;
    device_object->start_memory_copy_host = new hipEvent_t;
    device_object->stop_memory_copy_host= new hipEvent_t;
    
    hipEventCreate(device_object->start);
    hipEventCreate(device_object->stop);
    hipEventCreate(device_object->start_memory_copy_device);
    hipEventCreate(device_object->stop_memory_copy_device);
    hipEventCreate(device_object->start_memory_copy_host);
    hipEventCreate(device_object->stop_memory_copy_host);
}


bool device_memory_init(GraficObject *device_object, int64_t size_b_matrix){
    hipError_t err = hipSuccess;
    // Allocate the device input vector B
    err = hipMalloc((void **)&device_object->d_B, (size_b_matrix /2) * sizeof(bench_cuda_complex));

    if (err != hipSuccess)
    {
        return false;
    }
    return true;
}

void copy_memory_to_device(GraficObject *device_object, bench_t* h_B,int64_t size){
    hipError_t err = hipSuccess;
    bench_cuda_complex *h_signal = (bench_cuda_complex *)malloc(sizeof(bench_cuda_complex) * (size/2));
    for (unsigned int i = 0; i < (size/2); ++i){
        h_signal[i].x = h_B[i * 2];
        h_signal[i].y = h_B[i * 2 + 1];
    }

    hipEventRecord(*device_object->start_memory_copy_device);
    err = hipMemcpy(device_object->d_B, h_signal, sizeof(bench_cuda_complex) * (size / 2), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        return;
    }
    hipEventRecord(*device_object->stop_memory_copy_device);
    
}
void execute_kernel(GraficObject *device_object, int64_t size){

    hipfftHandle plan;

    hipEventRecord(*device_object->start);
    
    #ifdef FLOAT
    hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, (hipfftComplex *)device_object->d_B, (hipfftComplex *)device_object->d_B, HIPFFT_FORWARD);
    #else 
    hipfftPlan1d(&plan, size, HIPFFT_Z2Z, 1);
    hipfftExecZ2Z(plan, (hipfftDoubleComplex *)device_object->d_B, (hipfftDoubleComplex *)device_object->d_B, HIPFFT_FORWARD);
    #endif
    
    hipEventRecord(*device_object->stop);
    hipfftDestroy(plan);
    
}

void copy_memory_to_host(GraficObject *device_object, bench_t* h_B, int64_t size){
    bench_cuda_complex *h_signal = (bench_cuda_complex *)malloc(sizeof(bench_cuda_complex) * (size/2));
    hipEventRecord(*device_object->start_memory_copy_host);
    hipMemcpy(h_signal, device_object->d_B, (size/2) * sizeof(bench_cuda_complex), hipMemcpyDeviceToHost);
    hipEventRecord(*device_object->stop_memory_copy_host);

    for (unsigned int i = 0; i < (size/2); ++i){
        h_B[i * 2] = h_signal[i].x;
        h_B[i * 2 + 1] = h_signal[i].y;
    }
}

float get_elapsed_time(GraficObject *device_object, bool csv_format, bool csv_format_timestamp, long int current_time){
    hipEventSynchronize(*device_object->stop_memory_copy_host);
    float milliseconds_h_d = 0, milliseconds = 0, milliseconds_d_h = 0;
    // memory transfer time host-device
    hipEventElapsedTime(&milliseconds_h_d, *device_object->start_memory_copy_device, *device_object->stop_memory_copy_device);
    // kernel time
    hipEventElapsedTime(&milliseconds, *device_object->start, *device_object->stop);
    //  memory transfer time device-host
    hipEventElapsedTime(&milliseconds_d_h, *device_object->start_memory_copy_host, *device_object->stop_memory_copy_host);
    
    if (csv_format_timestamp){
        printf("%.10f;%.10f;%.10f;%ld;\n\n", milliseconds_h_d,milliseconds,milliseconds_d_h, current_time);
    }
    else if (csv_format){
         printf("%.10f;%.10f;%.10f;\n", milliseconds_h_d,milliseconds,milliseconds_d_h);
    }else{
         printf("Elapsed time Host->Device: %.10f milliseconds\n", milliseconds_h_d);
         printf("Elapsed time kernel: %.10f milliseconds\n", milliseconds);
         printf("Elapsed time Device->Host: %.10f milliseconds\n", milliseconds_d_h);
    }
    return milliseconds;
}

void clean(GraficObject *device_object){
    hipError_t err = hipSuccess;

    err = hipFree(device_object->d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return;
    }


    // delete events
    delete device_object->start;
    delete device_object->stop;
    delete device_object->start_memory_copy_device;
    delete device_object->stop_memory_copy_device;
    delete device_object->start_memory_copy_host;
    delete device_object->stop_memory_copy_host;
}